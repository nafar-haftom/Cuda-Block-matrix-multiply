#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE!
//YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "bmm.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

// TILEX and TILEY are used to set the number of threads in a CUDA block 
#define TILEX 16
#define TILEY 8

// you may define other parameters here!
// you may define other macros here!
// you may define other functions here!

dim3 getDimGrid(const int m, const int n) {
	dim3 dimGrid(n/TILEX,n/TILEY);
	return dimGrid;
}
dim3 getDimBlock(const int m, const int n) {
	dim3 dimBlock(TILEX,TILEY);
	return dimBlock;
}
__global__ void kernelFunc(float* ad, float* bd, float* cd, const int m, const int n) {

    int row = by * TILEY + ty;
    int col = bx * TILEX + tx;

    __shared__ float As[TILEY][TILEX*4];
    __shared__ float Bs[TILEY*4][TILEX];

    float Csub = 0.0;

    for (int p = 0 ; p < (n/TILEY) ; p = p + 4){
		
		//loop unrolling
		if(tx < TILEY){
			As[ty][tx] = ad[((row)<<(m)) + (TILEY*p+tx)];
		}
		if(tx < TILEY){
			As[ty][tx + TILEY] = ad[((row)<<(m)) + (TILEY*(p+1)+tx)];
		}
		if(tx < TILEY){
			As[ty][tx + 2*TILEY] = ad[((row)<<(m)) + (TILEY*(p+2)+tx)];
		}
		if(tx < TILEY){
			As[ty][tx + 3*TILEY] = ad[((row)<<(m)) + (TILEY*(p+3)+tx)];
		}
		//loop unrolling
		if (ty < TILEX){
			Bs[ty][tx] = bd[((TILEY*p + ty)<<(m)) + (col)];
		}
		if (ty < TILEX){
			Bs[ty + TILEY][tx] = bd[((TILEY * (p+1) + ty) << (m)) + (col)];
		}
		if (ty < TILEX){
			Bs[ty + 2*TILEY][tx] = bd[((TILEY * (p+2) + ty) << (m)) + (col)];
		}
		if (ty < TILEX){
			Bs[ty + 3*TILEY][tx] = bd[((TILEY * (p+3) + ty) << (m)) + (col)];
		}
		__syncthreads();
		
		for (int k = 0 ; k < 4 * TILEY ; k++){ 
			Csub += As[ ty ][ k ] * Bs[ k ][ tx ];
		}
		__syncthreads();
	}
	
	cd [( (row) << (m) ) + ( col )] = Csub;
}
